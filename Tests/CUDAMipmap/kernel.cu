#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include "hip/hip_fp16.h"
#include ""
#include <cstdio>
#include <cmath>
#include "CUDAMipmap.h"

#define uchar uint8_t
#define uchar8 half
#define ushort uint16_t
#define kernel __global__
#define restrict __restrict__
#define global  
#define constant  
#define local __shared__  
#define private  
#define barrier __syncthreads
#define CLK_LOCAL_MEM_FENCE 
inline __device__ float clamp(float f, float a, float b)
{
    return fmaxf(a, fminf(f, b));
}
inline __device__ float4 clamp(float4 v, float a, float b)
{
    return make_float4(clamp(v.x, a, b), clamp(v.y, a, b), clamp(v.z, a, b), clamp(v.w, a, b));
}
inline __device__ void operator*=(float4 &a, float s)
{
    a.x *= s; a.y *= s; a.z *= s; a.w *= s;
}
inline __device__ float4 operator*(float4 a, float s)
{
    return make_float4(a.x * s, a.y * s, a.z * s, a.w * s);
}
inline __device__ float4 operator*(float s, float4 a)
{
    return make_float4(a.x * s, a.y * s, a.z * s, a.w * s);
}
inline __device__ float4 operator+(float4 a, float4 b)
{
    return make_float4(a.x + b.x, a.y + b.y, a.z + b.z, a.w + b.w);
}
inline __device__ void operator+=(float4 &a, float4 b)
{
    a.x += b.x; a.y += b.y; a.z += b.z; a.w += b.w;
}
inline __device__ float4 operator*(uchar4 a, float4 b)
{
    return make_float4(a.x * b.x, a.y * b.y, a.z * b.z, a.w * b.w);
}
inline __device__ float4 operator*(float4 a, uchar4 b)
{
    return make_float4(a.x * b.x, a.y * b.y, a.z * b.z, a.w * b.w);
}
inline __device__ void vstorea_half4(float4 dat, size_t offset, half* __restrict__ ptr)
{
    (ptr + offset * 4)[0] = dat.x;
    (ptr + offset * 4)[1] = dat.x;
    (ptr + offset * 4)[2] = dat.x;
    (ptr + offset * 4)[3] = dat.x;
}
inline __device__ uchar4 convert_uchar4(float4 dat)
{
    return make_uchar4(dat.x, dat.y, dat.z, dat.w);
}


inline __device__ float SRGBToLinear(const float color)
{
    return color <= 0.04045f ? (1.0f / 12.92f) * color : pow((1.0f / 1.055f) * (color + 0.055f), 2.4f);
}
inline __device__ float4 SRGBToLinear(const float4 color)
{
    return make_float4(SRGBToLinear(color.x), SRGBToLinear(color.x), SRGBToLinear(color.x), color.w);
}
inline __device__ float LinearToSRGB(const float color)
{
    return color <= 0.00304f ? 12.92f * color : 1.055f * pow(color, 1.0f / 2.4f) - 0.055f;
}
inline __device__ float4 LinearToSRGB(const float4 color)
{
    return make_float4(LinearToSRGB(color.x), LinearToSRGB(color.x), LinearToSRGB(color.x), color.w);
}

typedef struct Info
{
    ushort SrcWidth;
    ushort SrcHeight;
    ushort LimitX;
    ushort LimitY;
}Info;



#define COEF_D1  0.5625f
#define COEF_D3 -0.0625f

#ifndef CountX
#   define CountX 8
#endif
#ifndef CountY
#   define CountY 8
#endif

__inline__ __device__ float4 loadU4_F4(const uchar4* ptr)
{
    const uchar4 dat = __ldg(ptr);
    return make_float4(dat.x, dat.y, dat.z, dat.w);
}

kernel void Downsample_Src(global const uchar4* restrict src, const Info* info, const uchar level, global uchar8* restrict mid, global uchar4* restrict dst)
{
    const ushort dstX = threadIdx.x + CountX * blockIdx.x, dstY = threadIdx.y + CountY * blockIdx.y;
    const uchar lidX = threadIdx.x, lidY = threadIdx.y, lid = lidY * CountX + lidX;
    local float4 sharedImg1[CountX*CountY], sharedImg2[CountX*CountY];

    const uchar* restrict ptrSrc = (global const uchar*)(src + (dstY * 4) * info[level].SrcWidth + (dstX * 4));

    float4 res[8];
#define LOOP_LINE(line) \
    { \
        float4 tmp1 = SRGBToLinear(loadU4_F4((const uchar4*)(ptrSrc   )) * (1.f/255.f)); \
        float4 tmp2 = SRGBToLinear(loadU4_F4((const uchar4*)(ptrSrc+ 4)) * (1.f/255.f)); \
        float4 tmp3 = SRGBToLinear(loadU4_F4((const uchar4*)(ptrSrc+ 8)) * (1.f/255.f)); \
        float4 tmp4 = SRGBToLinear(loadU4_F4((const uchar4*)(ptrSrc+12)) * (1.f/255.f)); \
        barrier(CLK_LOCAL_MEM_FENCE); \
        sharedImg1[lid] = tmp1; sharedImg2[lid] = tmp4; \
        barrier(CLK_LOCAL_MEM_FENCE); \
        float4 leftPix, rightPix; \
        if (lidX == 0) \
        { \
            if (dstX == 0) \
                leftPix = tmp1; \
            else \
                leftPix = SRGBToLinear(loadU4_F4((const uchar4*)(ptrSrc - 4)) * (1.f/255.f)); \
        } \
        else \
            leftPix = sharedImg2[lid - 1]; \
        if (lidX == CountX-1) \
        { \
            if (dstX == info[level].LimitX) \
                rightPix = tmp4; \
            else \
                rightPix = SRGBToLinear(loadU4_F4((const uchar4*)(ptrSrc + 16)) * (1.f/255.f)); \
        } \
        else \
            rightPix = sharedImg1[lid + 1]; \
        res[line*2  ] = clamp(leftPix   * COEF_D3 + tmp1 * COEF_D1 + tmp2 * COEF_D1 + tmp3      * COEF_D3, 0.f, 1.f); \
        res[line*2+1] = clamp(tmp2      * COEF_D3 + tmp3 * COEF_D1 + tmp4 * COEF_D1 + rightPix  * COEF_D3, 0.f, 1.f); \
        ptrSrc += info[level].SrcWidth * 4; \
    }
    LOOP_LINE(0)
    LOOP_LINE(1)
    LOOP_LINE(2)
    LOOP_LINE(3)
    {
        float4 upPix[2], downPix[2];
        if (lidY == 0)
        {
            if (dstY == 0)
            {
                upPix[0] = res[0]; upPix[1] = res[1];
            }
            else
            {
                float4 tmp1 = SRGBToLinear(loadU4_F4((const uchar4*)(ptrSrc - info[level].SrcWidth * 20)) * (1.f / 255.f));
                float4 tmp2 = SRGBToLinear(loadU4_F4((const uchar4*)(ptrSrc - info[level].SrcWidth * 16)) * (1.f / 255.f));
                float4 tmp3 = SRGBToLinear(loadU4_F4((const uchar4*)(ptrSrc - info[level].SrcWidth * 12)) * (1.f / 255.f));
                float4 tmp4 = SRGBToLinear(loadU4_F4((const uchar4*)(ptrSrc - info[level].SrcWidth *  8)) * (1.f / 255.f));
                upPix[0] = tmp1 * 0.5f + tmp2 * 0.5f; upPix[1] = tmp3 * 0.5f + tmp4 * 0.5f;
            }
        }
        else if (lidY == CountY - 1)
        {
            if (dstY == info[level].LimitY)
            {
                downPix[0] = res[6]; downPix[1] = res[7];
            }
            else
            {
                float4 tmp1 = SRGBToLinear(loadU4_F4((const uchar4*)(ptrSrc +  0)) * (1.f / 255.f));
                float4 tmp2 = SRGBToLinear(loadU4_F4((const uchar4*)(ptrSrc +  4)) * (1.f / 255.f));
                float4 tmp3 = SRGBToLinear(loadU4_F4((const uchar4*)(ptrSrc +  8)) * (1.f / 255.f));
                float4 tmp4 = SRGBToLinear(loadU4_F4((const uchar4*)(ptrSrc + 12)) * (1.f / 255.f));
                downPix[0] = tmp1 * 0.5f + tmp2 * 0.5f; downPix[1] = tmp3 * 0.5f + tmp4 * 0.5f;
            }
        }
        barrier(CLK_LOCAL_MEM_FENCE);
        sharedImg1[lid] = res[0]; sharedImg2[lid] = res[6];
        barrier(CLK_LOCAL_MEM_FENCE);
        if (lidY > 0)
            upPix[0] = sharedImg2[lid - CountX];
        if (lidY < CountY - 1)
            downPix[0] = sharedImg1[lid + CountX];
        barrier(CLK_LOCAL_MEM_FENCE);
        sharedImg1[lid] = res[1]; sharedImg2[lid] = res[7];
        barrier(CLK_LOCAL_MEM_FENCE);
        if (lidY > 0)
            upPix[1] = sharedImg2[lid - CountX];
        if (lidY < CountY - 1)
            downPix[1] = sharedImg1[lid + CountX];
        res[0] = clamp(upPix[0] * COEF_D3 + res[0] * COEF_D1 + res[2] * COEF_D1 + res[4] * COEF_D3, 0.f, 1.f);
        res[1] = clamp(upPix[1] * COEF_D3 + res[1] * COEF_D1 + res[3] * COEF_D1 + res[5] * COEF_D3, 0.f, 1.f);
        res[6] = clamp(res[2] * COEF_D3 + res[4] * COEF_D1 + res[6] * COEF_D1 + downPix[0] * COEF_D3, 0.f, 1.f);
        res[7] = clamp(res[3] * COEF_D3 + res[5] * COEF_D1 + res[7] * COEF_D1 + downPix[1] * COEF_D3, 0.f, 1.f);
    }
    global half* restrict ptrMid = (global half*)(mid + (dstY * 2) * info[level].SrcWidth / 2 + (dstX * 2));
    global uchar4* restrict ptrDst = (global uchar4*)(dst + (dstY * 2) * info[level].SrcWidth / 2 + (dstX * 2));
    vstorea_half4(res[0], 0, ptrMid); vstorea_half4(res[1], 1, ptrMid); 
    vstorea_half4(res[6], info[level].SrcWidth / 2, ptrMid); vstorea_half4(res[7], info[level].SrcWidth / 2 + 1, ptrMid);
    res[0] = LinearToSRGB(res[0]); res[1] = LinearToSRGB(res[1]);
    res[6] = LinearToSRGB(res[6]); res[7] = LinearToSRGB(res[7]);
    ptrDst[0] = convert_uchar4(res[0] * 255.0f); ptrDst[1] = convert_uchar4(res[1] * 255.0f);
    ptrDst[info[level].SrcWidth / 2] = convert_uchar4(res[6] * 255.0f); ptrDst[info[level].SrcWidth / 2 + 1] = convert_uchar4(res[7] * 255.0f);
}


#ifdef __cplusplus
extern "C" {
#endif
// Helper function for using CUDA to add vectors in parallel.
void CUDAMIPMAPAPI DoMipmap(const void* src, void* dst, const uint32_t width, const uint32_t height)
{
    uchar4 *dev_src = 0;
    uchar8 *dev_mid = 0;
    Info  *dev_info = 0;
    uchar4 *dev_dst = 0;
    hipError_t cudaStatus;
    Info info
    {
        static_cast<uint16_t>(width), static_cast<uint16_t>(height),
        static_cast<uint16_t>(width / 4 - 1),static_cast<uint16_t>(height / 4 - 1)
    };


    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) 
    {
        printf("hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_src, width*height * sizeof(uint32_t));
    if (cudaStatus != hipSuccess) 
    {
        printf("hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_info, sizeof(Info));
    if (cudaStatus != hipSuccess)
    {
        printf("hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_mid, width*height * sizeof(uint32_t));
    if (cudaStatus != hipSuccess) 
    {
        printf("hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_dst, width*height * sizeof(uint32_t) / 4);
    if (cudaStatus != hipSuccess) 
    {
        printf("hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_src, src, width*height * sizeof(uint32_t), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) 
    {
        printf("hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_info, &info, sizeof(Info), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        printf("hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    Downsample_Src<<<CountX, CountY>>>(dev_src, dev_info, 0, dev_mid, dev_dst);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) 
    {
        printf("addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) 
    {
        printf("hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    hipMemcpy(dst, dev_dst, width*height, hipMemcpyDeviceToHost);

Error:
    hipFree(dev_src);
    hipFree(dev_mid);
    hipFree(dev_dst);
    
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) 
    {
        printf("hipDeviceReset failed!");
    }
}

#ifdef __cplusplus
}
#endif

